#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <unistd.h>
#include <cmath>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../defs.h"
#include "cuda_global.h"
#include "cuda_defs.h"
#include "cuda_kernels.h"
#include "cuda_routines.h"


extern int _NNB;
int _NNB = 0;
static int nbodymax;
static int devid, numGPU;
static bool is_open = false;
static bool devinit = false;
static bool first   = true;
static int variable_size;


extern CUDA_REAL *h_ptcl, *d_ptcl; //, *background;
extern CUDA_REAL *h_result, *d_result;
extern CUDA_REAL *d_diff, *d_magnitudes, *d_r2;
extern int *h_neighbor, *d_neighbor, *h_num_neighbor, *d_num_neighbor;
extern int *d_target;

CUDA_REAL *h_ptcl=nullptr, *d_ptcl=nullptr;; //, *background;
CUDA_REAL *h_result=nullptr, *d_result=nullptr;
CUDA_REAL *d_diff=nullptr,*d_magnitudes=nullptr, *d_r2=nullptr;
int *h_neighbor=nullptr, *d_neighbor=nullptr, *d_num_neighbor=nullptr, *h_num_neighbor=nullptr;
int *d_target=nullptr;


extern hipStream_t stream;
hipStream_t stream;

extern CUDA_REAL *h_diff, *h_magnitudes;
CUDA_REAL *h_diff, *h_magnitudes;

/*************************************************************************
 *	 Computing Acceleration
 *************************************************************************/

void GetAcceleration(
		int NumTarget,
		int h_target_list[],
		CUDA_REAL acc[][3],
		CUDA_REAL adot[][3],
		int NumNeighbor[],
		int **NeighborList
		) {

	assert(is_open);
	assert((NumTarget > 0) && (NumTarget <= _NNB));

	int minGridSize, blockSize, gridSize;
	int sharedMemSize;

	//hipStreamCreate(&stream);

	hipblasHandle_t handle;
	initializeCudaAndCublas(&handle);

	/*
	for(int i=0; i<NumTarget; i++) {
		d_result[i].clear();
		d_neighbor[i].clear();
		d_dist = 0.;
	}
	*/
	/*
	fprintf(stderr,"\ntargets=");
	for(int i=0; i<NumTarget; i++) {
		fprintf(stderr,"%d, ", h_target_list[i]);
	}
	fprintf(stderr,"\n");
	*/


	//toDevice(h_target_list, d_target, NumTarget, stream);
	toDevice(h_target_list, d_target, NumTarget, stream);

	// Kernel launch parameters
	//dim3 blockSize(variable_size);
	//dim3 gridSize(NumTarget);
	//dim3 gridSize((NumTarget * NNB + blockSize.x - 1) / blockSize.x);

	// Compute pairwise differences for the subset

	//blockSize = variable_size;
	//gridSize = NumTarget;
	int total_data_num = new_size(_NNB*NumTarget);
	/******* Initialize *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	initialize, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	initialize<<<gridSize, blockSize, 0, stream>>>\
		(d_result, d_neighbor, d_num_neighbor, d_diff, d_magnitudes, _NNB, NumTarget, d_target);
	hipDeviceSynchronize();


	/******* Differencese *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_pairwise_diff_subset, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_pairwise_diff_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, _NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	/******* Magnitudes *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_magnitudes_subset, 0, 0));	
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_magnitudes_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_r2, d_diff, d_magnitudes, _NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	/******* Force *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	compute_forces_subset, 0, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	compute_forces_subset<<<gridSize, blockSize, 0, stream>>>\
		(d_ptcl, d_diff, d_magnitudes, _NNB, NumTarget, d_target);

	/******* Neighborhood *********/
#define MAX_SIZE 9
	sharedMemSize = ((MAX_SIZE+1)*512) * sizeof(int);
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
				assign_neighbor, sharedMemSize, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;

	
	//blockSize = std::min(blockSize, 512);
	//gridSize = (_NNB * NumTarget + blockSize - 1) / blockSize;

	//blockSize = variable_size;
	//gridSize = NumTarget;

	assign_neighbor<<<gridSize, blockSize, sharedMemSize, stream>>>\
		(d_neighbor, d_num_neighbor, d_r2, d_magnitudes, _NNB, NumTarget, d_target);
	hipDeviceSynchronize();

	/******* Reduction *********/
	checkCudaError(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,
			 	reduce_forces, 0, 0));
	gridSize = (total_data_num + blockSize - 1) / blockSize;
	//blockSize = _NNB;
	//gridSize  = NumTarget;
	//blockSize = 128;
	//blockSize = variable_size;
	//gridSize = NumTarget;


	//	sharedMemSize = 256 * sizeof(double);
	reduce_forces<<<gridSize, blockSize, 0, stream>>>\
		(d_diff, d_result, _NNB, NumTarget);
	hipDeviceSynchronize();

	/*
	print_forces_subset<<<gridSize, blockSize>>>\
		(d_result, NumTarget);
		*/




	/*
	toHost(h_diff, d_diff, _six*NumTarget*NNB);
	for (int i = 0; i < NumTarget; ++i) {
		//std::cerr << "PID=" << h_target_list[i] << std::endl;
		for (int j = 0; j < NNB; ++j) {
			std::cerr << h_diff[_six*(i * NNB + j)] << " ";
		}
		std::cerr << std::endl;
	}
	*/

	//toHost(h_result  , d_result  , variable_size, stream);
	//toHost(h_neighbor, d_neighbor, variable_size, stream);


	hipStreamSynchronize(stream); // Wait for all operations to finish

	toHost(h_result      , d_result      ,           _six*NumTarget);
	toHost(h_neighbor    , d_neighbor    , NumNeighborMax*NumTarget);
	toHost(h_num_neighbor, d_num_neighbor,                NumTarget);
	//printf("CUDA: transfer to host done\n");


	//hipStreamSynchronize(stream); // Wait for all operations to finish


	for (int i=0;i<NumTarget;i++) {
		for (int j=0;j<h_num_neighbor[i];j++) {
			NeighborList[i][j] = h_neighbor[NumNeighborMax*i+j];
		}
		NumNeighbor[i] = h_num_neighbor[i];

		/*
		fprintf(stderr, "%d (%d) neighbors of %d = ", i, h_target_list[i], h_num_neighbor[i]);
		for (int j=0;j<h_num_neighbor[i];j++) {
			fprintf(stderr, "%d, ", NeighborList[i][j]);
		}
		fprintf(stderr, "\n");
		*/

		/*
		fprintf(stderr, "PID=%d: a=(%.4e,%.4e,%.4e), adot=(%.4e,%.4e,%.4e)\n",
				h_target_list[i],
				h_result[_six*i],
				h_result[_six*i+1],
				h_result[_six*i+2],
				h_result[_six*i+3],
				h_result[_six*i+4],
				h_result[_six*i+5]
				);
				*/
	}
	//fprintf(stderr, "\n");

	// out data
	for (int i=0; i<NumTarget; i++) {
		acc[i][0]  = h_result[_six*i];
		acc[i][1]  = h_result[_six*i+1];
		acc[i][2]  = h_result[_six*i+2];
		adot[i][0] = h_result[_six*i+3];
		adot[i][1] = h_result[_six*i+4];
		adot[i][2] = h_result[_six*i+5];
	}

	hipblasDestroy(handle);
	/*
	my_free(h_background , d_background);
	my_free(h_result     , d_result);
	my_free(h_target     , d_target);
	my_free(h_neighbor   , d_neighbor);
	*/
	//hipStreamDestroy(stream);
	//my_free_d(do_neighbor);
	//printf("CUDA: done?\n");
}










/*************************************************************************
 *	 Communication with HOST
 *************************************************************************/
void _ReceiveFromHost(
		int __NNB,
		CUDA_REAL m[],
		CUDA_REAL x[][3],
		CUDA_REAL v[][3],
		CUDA_REAL r2[],
		CUDA_REAL mdot[]
		){

	nbodymax = 100000000;
	_NNB     = __NNB;  // this values can be different from NNB 
	assert(_NNB <= nbodymax);
	hipError_t cudaStatus;

	//printf("CUDA: receive starts\n");

	if ((first) || (new_size(_NNB) > variable_size )) {
		variable_size = new_size(_NNB);
		if (!first) {
			my_free(h_ptcl				 , d_ptcl);
			my_free(h_result       , d_result);
			my_free(h_neighbor     , d_neighbor);
			my_free(h_num_neighbor , d_num_neighbor);
			hipFree(d_target);
			hipFree(d_r2);
			hipFree(d_diff);
			hipFree(d_magnitudes);
		}
		else {
			first = false;
		}
		my_allocate(&h_ptcl         , &d_ptcl        ,         _seven*variable_size); // x,v,m
		my_allocate(&h_result       , &d_result      ,           _six*variable_size);
		my_allocate(&h_num_neighbor , &d_num_neighbor,                variable_size);
		my_allocate(&h_neighbor     , &d_neighbor    , NumNeighborMax*variable_size);
		hipMalloc((void**)&d_r2        ,        variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_target    ,        variable_size * sizeof(int));
		hipMalloc((void**)&d_diff      , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		hipMalloc((void**)&d_magnitudes, _two * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_diff          , _six * variable_size * variable_size * sizeof(CUDA_REAL));
		//hipHostMalloc((void**)&h_magnitudes    , _two * variable_size * variable_size * sizeof(CUDA_REAL));
	}


	for (int j=0; j<_NNB; j++) {
		for (int dim=0; dim<Dim; dim++) {
			h_ptcl[_seven*j+dim]   = x[j][dim];
			h_ptcl[_seven*j+dim+3] = v[j][dim];
		}
		h_ptcl[_seven*j+6] = m[j];
		//h_particle[j].setParticle(m[j], x[j], v[j], r2[j], mdot[j]);
	}

	//toDevice(h_background,d_background,variable_size);
	toDevice(h_ptcl,d_ptcl, _seven*_NNB, stream);
	toDevice(r2    ,d_r2  ,        _NNB, stream);
	//fprintf(stdout, "CUDA: receive done\n");
}





void _InitializeDevice() {

	if(is_open){
		fprintf(stderr, "it is already open\n");
		return;
	}
	is_open = true;

	std::cout << "Initializing CUDA ..." << std::endl;
	int device = 0; 
	int deviceCount;

	hipGetDeviceCount(&deviceCount);
	hipStreamCreate(&stream);

	std::cout << "There are " << deviceCount << " GPUs." << std::endl;


	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, devid);
	//  char *hostname = getenv("HOSTNAME");


	fprintf(stderr, "-------- GPU initialization : device: %s (%d)----------\n", prop.name, devid);


	hipSetDevice(device);


	is_open = true;

	std::cout << "CUDA initialized successfully!" << std::endl;
}



void _CloseDevice() {
	if(!is_open) {
		fprintf(stderr, "gpunb: it is already close\n");
		return;
	}
	is_open = false;


	hipError_t error;


	printf("CUDA: ?!! ...\n");
	//my_free(&h_result    , &d_result);
	fprintf(stderr, "result ...\n");
	//my_free(&h_target    , &d_target);
	fprintf(stderr, "target ...\n");
	//my_free(&h_neighbor  , &d_neighbor);
	fprintf(stderr, "neighbor ...\n");
	//my_free(&h_background, &d_background);

}





extern "C" {
	void InitializeDevice() {
		_InitializeDevice();
	}
	void CloseDevice(){
		_CloseDevice();
	}
	void SendToDevice(int *__NNB, CUDA_REAL m[], CUDA_REAL x[][3], CUDA_REAL v[][3], CUDA_REAL r2[], CUDA_REAL mdot[]) {
		_ReceiveFromHost(*__NNB, m, x, v, r2, mdot);
	}
	void CalculateAccelerationOnDevice(int *NumTarget, int *h_target_list, CUDA_REAL acc[][3], CUDA_REAL adot[][3], int NumNeighbor[], int **NeighborList) {
		GetAcceleration(*NumTarget, h_target_list, acc, adot, NumNeighbor, NeighborList);
	}
}

